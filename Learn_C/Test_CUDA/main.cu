
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel to add two vectors on GPU
__global__
void addVectors(int *a, int *b, int *c, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int size = 1024; // Vector size
    int *h_a, *h_b, *h_c; // Host vectors
    int *d_a, *d_b, *d_c; // Device vectors

    // Allocate memory on host and device
    h_a = new int[size];
    h_b = new int[size];
    h_c = new int[size];
    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMalloc((void**)&d_c, size * sizeof(int));

    // Initialize host vectors
    for (int i = 0; i < size; ++i) {
        h_a[i] = i;
        h_b[i] = size - i;
    }

    // Copy host vectors to device
    hipMemcpy(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with 1 block and 'size' threads
    addVectors<<<1, size>>>(d_a, d_b, d_c, size);

    // Copy result from device to host
    hipMemcpy(h_c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    for (int i = 0; i < size; ++i) {
        std::cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << std::endl;
    }

    // Clean up
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
